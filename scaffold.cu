

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime_api.h>


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest();
void foo();
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main() 
{
    runTest();

}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest() 
{

    hipSetDevice(1);
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    float milliseconds = -1;
    checkCudaErrors(hipEventRecord(start));
    foo();
    checkCudaErrors(hipEventRecord(stop));
    hipDeviceSynchronize();
    milliseconds = -1;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    //printf("GPU Processing time: %f (ms)\n", milliseconds);
    printf("%f", milliseconds);
}
